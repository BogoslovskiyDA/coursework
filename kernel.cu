#include <stdio.h>
#include <hip/hip_runtime.h>

#define dx 0.01
#define dy 0.01
#define rho 8800
#define C 381
#define lambda 384.0
#define tau 0.01
#define BLOCK_SIZE 16

__global__ void __laplas__(float *T,float *T_old, const int n, const int height)
{
        double at = lambda / (rho * C);

        int iA = n * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;

        if(blockDim.y * blockIdx.y + threadIdx.y>0 && blockDim.x * blockIdx.x + threadIdx.x>0 && blockDim.y * blockIdx.y + threadIdx.y < height && blockDim.x * blockIdx.x + threadIdx.x < n)
                T[iA] = T_old[iA] + (tau / (dx * dx)) * at * (T_old[n * (blockDim.y * blockIdx.y + threadIdx.y-1) + blockDim.x * blockIdx.x + threadIdx.x] - 2 * T_old[iA] + T_old[n * (blockDim.y * blockIdx.y + threadIdx.y+1) + blockDim.x * blockIdx.x + threadIdx.x]) + (tau / (dy * dy)) * at * (T_old[n * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x-1] - 2 * T_old[iA] + T_old[n * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x+1]);
}

extern "C" void gpu(int index, int numDev, int n, int height, float *T, float *T_old)
{
        hipSetDevice(numDev);
        if (index == 0)
        {
                int count;
                unsigned int flag;
                int device;
                hipGetDevice(&device);
                hipGetDeviceCount(&count);
                hipGetDeviceFlags(&flag);
                printf("set device %d\n", numDev);
                printf("device %d\n", device);
                printf("device flag %d\n", flag);
                printf("device count %d\n", count);
        }
        size_t size = (height+1) * (n+1) * sizeof(float);

        float *dev_T = NULL;
        hipMalloc((void **)&dev_T, size);
        float *dev_T_old = NULL;
        hipMalloc((void **)&dev_T_old, size);

        hipMemcpy( dev_T, T, size, hipMemcpyHostToDevice );
        hipMemcpy( dev_T_old, T_old, size, hipMemcpyHostToDevice );

        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
        dim3 dimGrid(n/BLOCK_SIZE, height/BLOCK_SIZE, 1);

        __laplas__<<<dimGrid, dimBlock>>>(dev_T,dev_T_old, n, height);
        hipDeviceSynchronize();

        hipMemcpy(T, dev_T, size, hipMemcpyDeviceToHost);

        hipFree(dev_T);
        hipFree(dev_T_old);
}
